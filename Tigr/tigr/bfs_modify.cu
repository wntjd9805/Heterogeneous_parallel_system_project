#include "hip/hip_runtime.h"

#include "../shared/timer.hpp"
#include "../shared/tigr_utilities.hpp"
#include "../shared/graph.hpp"
#include "../shared/virtual_graph.hpp"
#include "../shared/globals.hpp"
#include "../shared/argument_parsing.hpp"
#include "../shared/gpu_error_check.cuh"




__global__ void kernel(unsigned int numParts, 
							unsigned int *nodePointer, 
							PartPointer *partNodePointer,
							unsigned int *edgeList, 
							unsigned int *dist, 
							bool *finished,
							int level)
{
	unsigned int partId = blockDim.x * blockIdx.x + threadIdx.x;

	if(partId < numParts)
	{
		unsigned int id = partNodePointer[partId].node;
		unsigned int part = partNodePointer[partId].part;

		if(dist[id] != level)
			return;

		unsigned int thisPointer = nodePointer[id];
		unsigned int degree = edgeList[thisPointer];
			
		unsigned int numParts;
		if(degree % Part_Size == 0)
			numParts = degree / Part_Size ;
		else
			numParts = degree / Part_Size + 1;

		
		unsigned int end;

		unsigned int ofs = thisPointer + part + 1;

		for(int i=0; i<Part_Size; i++)
		{
			if(part + i*numParts >= degree)
				break;
			end = ofs + i*numParts;
			
			if(dist[edgeList[end]] == DIST_INFINITY)
			{
				dist[edgeList[end]] = level + 1;
				*finished = false;
			}
		}
		
	}
}


__global__ void clearLabel(bool *label, unsigned int size)
{
	unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
	if(id < size)
		label[id] = false;
}

__global__ void unifyMaxValues(unsigned int* d_dist_s, unsigned int* d_dist_d, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        d_dist_s[idx] = max(d_dist_s[idx], d_dist_d[idx]);
        d_dist_d[idx] = d_dist_s[idx]; // This line ensures both arrays have the same value.
    }
}
int main(int argc, char** argv)
{	
	ArgumentParser arguments(argc, argv, true, false);
	
	Graph graph_d("/root/Tigr/datasets/LiveJournal/dense.txt", false);
	Graph graph_s("/root/Tigr/datasets/LiveJournal/sparse.txt", false);
	
	graph_d.ReadGraph();
    graph_s.ReadGraph();
	
	VirtualGraph vGraph_d(graph_d);
	VirtualGraph vGraph_s(graph_s);
	vGraph_d.MakeUGraph();
	vGraph_s.MakeUGraph();
	
	uint num_nodes_d = graph_d.num_nodes;
	uint num_edges_d = graph_d.num_edges;

	uint num_nodes_s = graph_s.num_nodes;
	uint num_edges_s = graph_s.num_edges;
	
	if(arguments.hasDeviceID)
		hipSetDevice(arguments.deviceID);

	hipFree(0);
	
	unsigned int *dist_d;
	dist_d  = new unsigned int[num_nodes_d];

	for(int i=0; i<num_nodes_d; i++)
	{
		dist_d[i] = DIST_INFINITY;
	}
	dist_d[arguments.sourceNode] = 0;

	unsigned int *dist_s;
	dist_s  = new unsigned int[num_nodes_s];

	for(int i=0; i<num_nodes_s; i++)
	{
		dist_s[i] = DIST_INFINITY;
	}
	dist_s[arguments.sourceNode] = 0;
	

	unsigned int *d_nodePointer_d, *d_edgeList_d, *d_dist_d;
    PartPointer *d_partNodePointer_d; 
    bool *d_finished_d;
    // graph_d에 대한 메모리 할당
    gpuErrorcheck(hipMalloc(&d_nodePointer_d, num_nodes_d * sizeof(unsigned int)));
    gpuErrorcheck(hipMalloc(&d_edgeList_d, (num_edges_d + num_nodes_d) * sizeof(unsigned int)));
    gpuErrorcheck(hipMalloc(&d_dist_d, num_nodes_d * sizeof(unsigned int)));
    gpuErrorcheck(hipMalloc(&d_finished_d, sizeof(bool)));
    gpuErrorcheck(hipMalloc(&d_partNodePointer_d, vGraph_d.numParts * sizeof(PartPointer)));

	gpuErrorcheck(hipMemcpy(d_nodePointer_d, vGraph_d.nodePointer, num_nodes_d * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_edgeList_d, vGraph_d.edgeList, (num_edges_d + num_nodes_d) * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_dist_d, dist_d, num_nodes_d * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_partNodePointer_d, vGraph_d.partNodePointer, vGraph_d.numParts * sizeof(PartPointer), hipMemcpyHostToDevice));
	
	unsigned int *d_nodePointer_s, *d_edgeList_s, *d_dist_s;
    PartPointer *d_partNodePointer_s; 
    bool *d_finished_s;
    // graph_s에 대한 메모리 할당
    gpuErrorcheck(hipMalloc(&d_nodePointer_s, num_nodes_s * sizeof(unsigned int)));
    gpuErrorcheck(hipMalloc(&d_edgeList_s, (num_edges_s + num_nodes_s) * sizeof(unsigned int)));
    gpuErrorcheck(hipMalloc(&d_dist_s, num_nodes_s * sizeof(unsigned int)));
    gpuErrorcheck(hipMalloc(&d_finished_s, sizeof(bool)));
    gpuErrorcheck(hipMalloc(&d_partNodePointer_s, vGraph_s.numParts * sizeof(PartPointer)));
	
	gpuErrorcheck(hipMemcpy(d_nodePointer_s, vGraph_s.nodePointer, num_nodes_s * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_edgeList_s, vGraph_s.edgeList, (num_edges_s + num_nodes_s) * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_dist_s, dist_s, num_nodes_s * sizeof(unsigned int), hipMemcpyHostToDevice));
	gpuErrorcheck(hipMemcpy(d_partNodePointer_s, vGraph_s.partNodePointer, vGraph_s.numParts * sizeof(PartPointer), hipMemcpyHostToDevice));
	
	hipStream_t stream1, stream2;
	hipStreamCreate(&stream1);
	hipStreamCreate(&stream2);
	bool finished_d;
	bool finished_s;

	Timer t;
	t.Start();

	int itr = 0;
	int level = 0;
	do
	{
		itr++;
		finished_d=true;
		finished_s = true;
		gpuErrorcheck(hipMemcpy(d_finished_d, &finished_d, sizeof(bool), hipMemcpyHostToDevice));
		gpuErrorcheck(hipMemcpy(d_finished_s, &finished_s, sizeof(bool), hipMemcpyHostToDevice));
		if(itr % 2 == 1)
		{
			kernel<<< vGraph_d.numParts/512 + 1 , 512,0, stream1  >>>(vGraph_d.numParts, 
														d_nodePointer_d,
														d_partNodePointer_d,
														d_edgeList_d, 
														d_dist_d, 
														d_finished_d,
														level);
			kernel<<< vGraph_s.numParts/512 + 1 , 512,0, stream2 >>>(vGraph_s.numParts, 
														d_nodePointer_s,
														d_partNodePointer_s,
														d_edgeList_s, 
														d_dist_s, 
														d_finished_s,
														level);
			// Assuming 'size' is the size of your d_dist_s and d_dist_d arrays
			int threadsPerBlock = 512;
			int blocksPerGrid = (num_nodes_d + threadsPerBlock - 1) / threadsPerBlock;
			unifyMaxValues<<<blocksPerGrid, threadsPerBlock,0, stream2>>>(d_dist_s, d_dist_d, num_nodes_d);

		}
		else
		{
			kernel<<< vGraph_d.numParts/512 + 1 , 512,0, stream1>>>(vGraph_d.numParts, 
														d_nodePointer_d, 
														d_partNodePointer_d,
														d_edgeList_d, 
														d_dist_d, 
														d_finished_d,
														level);													
			kernel<<< vGraph_s.numParts/512 + 1 , 512 ,0, stream2>>>(vGraph_s.numParts, 
														d_nodePointer_s, 
														d_partNodePointer_s,
														d_edgeList_s, 
														d_dist_s, 
														d_finished_s,
														level);		
			int threadsPerBlock = 512;
			int blocksPerGrid = (num_nodes_d + threadsPerBlock - 1) / threadsPerBlock;
			unifyMaxValues<<<blocksPerGrid, threadsPerBlock,0, stream2>>>(d_dist_s, d_dist_d, num_nodes_d);
		}
	
		gpuErrorcheck( hipPeekAtLastError() );
		gpuErrorcheck( hipDeviceSynchronize() );
		
		gpuErrorcheck(hipMemcpy(&finished_d, d_finished_d, sizeof(bool), hipMemcpyDeviceToHost));
		gpuErrorcheck(hipMemcpy(&finished_s, d_finished_s, sizeof(bool), hipMemcpyDeviceToHost));
		level++;

	} while (!(finished_d));
	
	cout << "Number of iterations = " << itr << endl;

	float runtime = t.Finish();
	cout << "Processing finished in " << runtime << " (ms).\n";
		
	
	gpuErrorcheck(hipMemcpy(dist_d, d_dist_d, num_nodes_d*sizeof(unsigned int), hipMemcpyDeviceToHost));
	gpuErrorcheck(hipMemcpy(dist_s, d_dist_s, num_nodes_s*sizeof(unsigned int), hipMemcpyDeviceToHost));
	utilities::PrintResults(dist_d, 30);
	utilities::PrintResults(dist_s, 30);
	if(arguments.hasOutput)
		utilities::SaveResults(arguments.output, dist_d, num_nodes_d);
		utilities::SaveResults(arguments.output, dist_s, num_nodes_s);
	

	gpuErrorcheck(hipFree(d_nodePointer_d));
    gpuErrorcheck(hipFree(d_edgeList_d));
    gpuErrorcheck(hipFree(d_dist_d));
    gpuErrorcheck(hipFree(d_finished_d));
    gpuErrorcheck(hipFree(d_partNodePointer_d));

    gpuErrorcheck(hipFree(d_nodePointer_s));
    gpuErrorcheck(hipFree(d_edgeList_s));
    gpuErrorcheck(hipFree(d_dist_s));
    gpuErrorcheck(hipFree(d_finished_s));
    gpuErrorcheck(hipFree(d_partNodePointer_s));

    delete[] dist_d;
    delete[] dist_s;
	return 0;

}
